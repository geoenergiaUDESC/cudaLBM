#include "LBMIncludes.cuh"
#include "LBMTypedefs.cuh"
#include "momentBasedD3Q19.cuh"
#include "fileIO/fileIO.cuh"
#include "runTimeIO/runTimeIO.cuh"
#include "postProcess.cuh"
#include "fieldAverage.cuh"

using namespace LBM;

[[nodiscard]] const std::array<hipStream_t, 1> createCudaStream() noexcept
{
    std::array<hipStream_t, 1> streamsLBM;

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipStreamCreate(&streamsLBM[0]));
    checkCudaErrors(hipDeviceSynchronize());

    return streamsLBM;
}

int main(int argc, char *argv[])
{
    const programControl programCtrl(argc, argv);

    const host::latticeMesh mesh;

    VelocitySet::D3Q19::print();

    const host::array<scalar_t, ctorType::READ_IF_PRESENT> hostMoments(programCtrl, mesh);

    // Set cuda device
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipSetDevice(programCtrl.deviceList()[0]));
    checkCudaErrors(hipDeviceSynchronize());

    // Setup Streams
    const std::array<hipStream_t, 1> streamsLBM = createCudaStream();

    // Perform device memory allocation
    device::array<scalar_t> deviceMoments(
        hostMoments.arr(),
        {"rho", "u", "v", "w", "m_xx", "m_xy", "m_xz", "m_yy", "m_yz", "m_zz"},
        mesh);
    device::halo blockHalo(hostMoments.arr(), mesh);
    // const device::array<nodeType_t> nodeTypes(host::nodeType(mesh), {"nodeTypes"}, mesh);

    // Set up time averaging
    // device::array<scalar_t> momentsMean(
    //     host::moments(mesh, programCtrl.u_inf()),
    //     {"rhoMean", "uMean", "vMean", "wMean", "m_xxMean", "m_xyMean", "m_xzMean", "m_yyMean", "m_yzMean", "m_zzMean"},
    //     mesh);

    // Copy symbols to device
    mesh.copyDeviceSymbols();
    programCtrl.copyDeviceSymbols(mesh.nx());

    std::cout << "Time loop start" << std::endl;
    std::cout << std::endl;

    const std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();

    for (label_t timeStep = programCtrl.latestTime(); timeStep < programCtrl.nt(); timeStep++)
    {
        if (programCtrl.print(timeStep))
        {
            std::cout << "Time: " << timeStep << "\n";
        }

        momentBasedD3Q19<<<mesh.gridBlock(), mesh.threadBlock(), 0, streamsLBM[0]>>>(
            deviceMoments.ptr(),
            blockHalo);

        // checkCudaErrors(hipDeviceSynchronize());
        // fieldAverage::calculate<<<mesh.gridBlock(), mesh.threadBlock(), 0, streamsLBM[0]>>>(
        //     deviceMoments.ptr(),
        //     momentsMean.ptr(),
        //     nodeTypes.ptr(),
        //     timeStep);

        blockHalo.swap();

        if (programCtrl.save(timeStep))
        {
            deviceMoments.write(programCtrl.caseName(), timeStep);

            if (timeStep > 0)
            {
                postProcess::writeTecplotHexahedralData(
                    fileIO::deinterleaveAoS(host::copyToHost(deviceMoments.ptr(), deviceMoments.size()), mesh),
                    programCtrl.caseName() + "_" + std::to_string(timeStep) + ".dat",
                    mesh,
                    deviceMoments.varNames(),
                    "Title");
            }

            // momentsMean.write(programCtrl.caseName(), timeStep);

            // postProcess::writeTecplotHexahedralData(
            //     fileIO::deinterleaveAoS(host::copyToHost(momentsMean.ptr(), mesh.nPoints() * 10), mesh),
            //     programCtrl.caseName() + "Mean_" + std::to_string(timeStep) + ".dat",
            //     mesh,
            //     momentsMean.varNames(),
            //     "Title");
        }

        // checkCudaErrors(hipDeviceSynchronize());
    }

    // Get ending time point and output the elapsed time
    const std::chrono::high_resolution_clock::time_point end = std::chrono::high_resolution_clock::now();
    std::cout << std::endl;
    std::cout << "Elapsed time: " << runTimeIO::duration(std::chrono::duration_cast<std::chrono::seconds>(end - start).count()) << std::endl;
    std::cout << std::endl;
    std::cout << "MLUPS: " << runTimeIO::MLUPS<double>(mesh, programCtrl, start, end) << std::endl;
    std::cout << "End" << std::endl;

    return 0;
}