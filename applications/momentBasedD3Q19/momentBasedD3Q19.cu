/*---------------------------------------------------------------------------*\
|                                                                             |
| cudaLBM: CUDA-based moment representation Lattice Boltzmann Method          |
| Developed at UDESC - State University of Santa Catarina                     |
| Website: https://www.udesc.br                                               |
| Github: https://github.com/geoenergiaUDESC/cudaLBM                          |
|                                                                             |
\*---------------------------------------------------------------------------*/

/*---------------------------------------------------------------------------*\

Copyright (C) 2023 UDESC Geoenergia Lab
Authors: Nathan Duggins (Geoenergia Lab, UDESC)

This implementation is derived from concepts and algorithms developed in:
  MR-LBM: Moment Representation Lattice Boltzmann Method
  Copyright (C) 2021 CERNN
  Developed at Universidade Federal do Paraná (UFPR)
  Original authors: V. M. de Oliveira, M. A. de Souza, R. F. de Souza
  GitHub: https://github.com/CERNN/MR-LBM
  Licensed under GNU General Public License version 2

License
    This file is part of cudaLBM.

    cudaLBM is free software: you can redistribute it and/or modify it
    under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.

Description
    Implementation of the moment representation with the D3Q19 velocity set

Namespace
    LBM

SourceFiles
    momentBasedD3Q19.cu

\*---------------------------------------------------------------------------*/

#include "momentBasedD3Q19.cuh"

using namespace LBM;

constexpr const label_t NStreams = 4;

int main(const int argc, const char *const argv[])
{
    const programControl programCtrl(argc, argv);

    // Set cuda device
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipSetDevice(programCtrl.deviceList()[0]));
    checkCudaErrors(hipDeviceSynchronize());

    const host::latticeMesh mesh(programCtrl);

    VelocitySet::print();

    // Allocate the arrays on the host first
    const host::array<scalar_t, VelocitySet> h_rho("rho", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_u("u", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_v("v", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_w("w", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_m_xx("m_xx", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_m_xy("m_xy", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_m_xz("m_xz", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_m_yy("m_yy", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_m_yz("m_yz", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_m_zz("m_zz", mesh, programCtrl);

    device::array<scalar_t> rho(h_rho, mesh);
    device::array<scalar_t> u(h_u, mesh);
    device::array<scalar_t> v(h_v, mesh);
    device::array<scalar_t> w(h_w, mesh);
    device::array<scalar_t> mxx(h_m_xx, mesh);
    device::array<scalar_t> mxy(h_m_xy, mesh);
    device::array<scalar_t> mxz(h_m_xz, mesh);
    device::array<scalar_t> myy(h_m_yy, mesh);
    device::array<scalar_t> myz(h_m_yz, mesh);
    device::array<scalar_t> mzz(h_m_zz, mesh);

    const device::ptrCollection<10, scalar_t> devPtrs(
        rho.ptr(),
        u.ptr(),
        v.ptr(),
        w.ptr(),
        mxx.ptr(),
        mxy.ptr(),
        mxz.ptr(),
        myy.ptr(),
        myz.ptr(),
        mzz.ptr());

    device::halo<VelocitySet> blockHalo(
        {h_rho.arr(),
         h_u.arr(),
         h_v.arr(),
         h_w.arr(),
         h_m_xx.arr(),
         h_m_xy.arr(),
         h_m_xz.arr(),
         h_m_yy.arr(),
         h_m_yz.arr(),
         h_m_zz.arr()},
        mesh);

    // Setup Streams
    const std::array<hipStream_t, NStreams> streamsLBM = host::createCudaStreams<NStreams>();

    const label_t z_stream_segment_size = mesh.nz() / NStreams;

    const dim3 blockDimensions{
        static_cast<uint32_t>(mesh.nx() / block::nx()),
        static_cast<uint32_t>(mesh.ny() / block::ny()),
        static_cast<uint32_t>(mesh.nz() / (NStreams * block::nz()))};

    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(momentBasedD3Q19), hipFuncCachePreferShared));

    const runTimeIO IO(mesh, programCtrl);

    for (label_t timeStep = programCtrl.latestTime(); timeStep < programCtrl.nt(); timeStep++)
    {
        // Do the run-time IO
        if (programCtrl.print(timeStep))
        {
            std::cout << "Time: " << timeStep << std::endl;
        }

        // Checkpoint
        if (programCtrl.save(timeStep))
        {
            fileIO::writeFile(
                programCtrl.caseName() + "_" + std::to_string(timeStep) + ".LBMBin",
                mesh,
                {"rho", "u", "v", "w", "m_xx", "m_xy", "m_xz", "m_yy", "m_yz", "m_zz"},
                host::toHost(devPtrs, mesh),
                timeStep);
        }

        // Main kernel
        for (label_t stream = 0; stream < NStreams; stream++)
        {
            momentBasedD3Q19<<<blockDimensions, mesh.threadBlock(), 0, streamsLBM[stream]>>>(
                devPtrs,
                blockHalo.fGhost(),
                blockHalo.gGhost(),
                z_stream_segment_size * stream);
        }

        for (label_t stream = 0; stream < NStreams; stream++)
        {
            hipStreamSynchronize(streamsLBM[stream]);
        }

        // Halo pointer swap
        blockHalo.swap();
    }

    for (label_t stream = 0; stream < NStreams; stream++)
    {
        hipStreamDestroy(streamsLBM[stream]);
    }

    return 0;
}