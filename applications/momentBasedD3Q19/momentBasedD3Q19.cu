/*---------------------------------------------------------------------------*\
|                                                                             |
| cudaLBM: CUDA-based moment representation Lattice Boltzmann Method          |
| Developed at UDESC - State University of Santa Catarina                     |
| Website: https://www.udesc.br                                               |
| Github: https://github.com/geoenergiaUDESC/cudaLBM                          |
|                                                                             |
\*---------------------------------------------------------------------------*/

/*---------------------------------------------------------------------------*\

Copyright (C) 2023 UDESC Geoenergia Lab
Authors: Nathan Duggins (Geoenergia Lab, UDESC)

This implementation is derived from concepts and algorithms developed in:
  MR-LBM: Moment Representation Lattice Boltzmann Method
  Copyright (C) 2021 CERNN
  Developed at Universidade Federal do Paraná (UFPR)
  Original authors: V. M. de Oliveira, M. A. de Souza, R. F. de Souza
  GitHub: https://github.com/CERNN/MR-LBM
  Licensed under GNU General Public License version 2

License
    This file is part of cudaLBM.

    cudaLBM is free software: you can redistribute it and/or modify it
    under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.

Description
    Implementation of the moment representation with the D3Q19 velocity set

Namespace
    LBM

SourceFiles
    momentBasedD3Q19.cu

\*---------------------------------------------------------------------------*/

#include "momentBasedD3Q19.cuh"

using namespace LBM;

__host__ [[nodiscard]] inline consteval label_t NStreams() noexcept { return 1; }

int main(const int argc, const char *const argv[])
{
    const programControl programCtrl(argc, argv);

    // Set cuda device
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipSetDevice(programCtrl.deviceList()[0]));
    checkCudaErrors(hipDeviceSynchronize());

    const host::latticeMesh mesh(programCtrl);

    VelocitySet::print();

    // Allocate the arrays on the device
    device::array<scalar_t, VelocitySet, time::instantaneous> rho("rho", mesh, programCtrl);
    device::array<scalar_t, VelocitySet, time::instantaneous> u("u", mesh, programCtrl);
    device::array<scalar_t, VelocitySet, time::instantaneous> v("v", mesh, programCtrl);
    device::array<scalar_t, VelocitySet, time::instantaneous> w("w", mesh, programCtrl);
    device::array<scalar_t, VelocitySet, time::instantaneous> mxx("m_xx", mesh, programCtrl);
    device::array<scalar_t, VelocitySet, time::instantaneous> mxy("m_xy", mesh, programCtrl);
    device::array<scalar_t, VelocitySet, time::instantaneous> mxz("m_xz", mesh, programCtrl);
    device::array<scalar_t, VelocitySet, time::instantaneous> myy("m_yy", mesh, programCtrl);
    device::array<scalar_t, VelocitySet, time::instantaneous> myz("m_yz", mesh, programCtrl);
    device::array<scalar_t, VelocitySet, time::instantaneous> mzz("m_zz", mesh, programCtrl);

    const device::ptrCollection<10, scalar_t> devPtrs(
        rho.ptr(),
        u.ptr(),
        v.ptr(),
        w.ptr(),
        mxx.ptr(),
        mxy.ptr(),
        mxz.ptr(),
        myy.ptr(),
        myz.ptr(),
        mzz.ptr());

    // Setup Streams
    const streamHandler<NStreams()> streamsLBM;

    objectRegistry<VelocitySet, NStreams()> runTimeObjects(mesh, devPtrs, streamsLBM);

    device::halo<VelocitySet> blockHalo(mesh, programCtrl);

    const device::ptrCollection<6, scalar_t> SPtrs(
        runTimeObjects.S().xx(), runTimeObjects.S().xy(),
        runTimeObjects.S().xz(), runTimeObjects.S().yy(),
        runTimeObjects.S().yz(), runTimeObjects.S().zz());
    const device::ptrCollection<6, scalar_t> SMeanPtrs(
        runTimeObjects.S().xxMean(), runTimeObjects.S().xyMean(),
        runTimeObjects.S().xzMean(), runTimeObjects.S().yyMean(),
        runTimeObjects.S().yzMean(), runTimeObjects.S().zzMean());

    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(momentBasedD3Q19), hipFuncCachePreferShared));

    const runTimeIO IO(mesh, programCtrl);

    for (label_t timeStep = programCtrl.latestTime(); timeStep < programCtrl.nt(); timeStep++)
    {
        // Do the run-time IO
        if (programCtrl.print(timeStep))
        {
            std::cout << "Time: " << timeStep << std::endl;
        }

        // Checkpoint
        if (programCtrl.save(timeStep))
        {
            fileIO::writeFile<time::instantaneous>(
                programCtrl.caseName() + "_" + std::to_string(timeStep) + ".LBMBin",
                mesh,
                functionObjects::solutionVariableNames,
                host::toHost(devPtrs, mesh),
                timeStep);

            if (runTimeObjects.S().calculate())
            {
                fileIO::writeFile<time::instantaneous>(
                    runTimeObjects.S().fieldName() + "_" + std::to_string(timeStep) + ".LBMBin",
                    mesh,
                    runTimeObjects.S().componentNames(),
                    host::toHost(SPtrs, mesh),
                    timeStep);
            }

            if (runTimeObjects.S().calculateMean())
            {
                fileIO::writeFile<time::timeAverage>(
                    runTimeObjects.S().fieldNameMean() + "_" + std::to_string(timeStep) + ".LBMBin",
                    mesh,
                    runTimeObjects.S().componentNamesMean(),
                    host::toHost(SMeanPtrs, mesh),
                    timeStep);
            }
        }

        // Main kernel
        host::constexpr_for<0, NStreams()>(
            [&](const auto stream)
            {
                momentBasedD3Q19<<<mesh.gridBlock(), mesh.threadBlock(), 0, streamsLBM.streams()[stream]>>>(devPtrs, blockHalo.fGhost(), blockHalo.gGhost());
            });

        // Calculate S kernel
        runTimeObjects.calculate(timeStep);

        // Halo pointer swap
        blockHalo.swap();
    }

    return 0;
}