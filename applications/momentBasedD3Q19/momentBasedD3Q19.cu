/*---------------------------------------------------------------------------*\
|                                                                             |
| cudaLBM: CUDA-based moment representation Lattice Boltzmann Method          |
| Developed at UDESC - State University of Santa Catarina                     |
| Website: https://www.udesc.br                                               |
| Github: https://github.com/geoenergiaUDESC/cudaLBM                          |
|                                                                             |
\*---------------------------------------------------------------------------*/

/*---------------------------------------------------------------------------*\

Copyright (C) 2023 UDESC Geoenergia Lab
Authors: Nathan Duggins (Geoenergia Lab, UDESC)

This implementation is derived from concepts and algorithms developed in:
  MR-LBM: Moment Representation Lattice Boltzmann Method
  Copyright (C) 2021 CERNN
  Developed at Universidade Federal do Paraná (UFPR)
  Original authors: V. M. de Oliveira, M. A. de Souza, R. F. de Souza
  GitHub: https://github.com/CERNN/MR-LBM
  Licensed under GNU General Public License version 2

License
    This file is part of cudaLBM.

    cudaLBM is free software: you can redistribute it and/or modify it
    under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.

Description
    Implementation of the moment representation with the D3Q19 velocity set

Namespace
    LBM

SourceFiles
    momentBasedD3Q19.cu

\*---------------------------------------------------------------------------*/

#include "momentBasedD3Q19.cuh"

using namespace LBM;

int main(const int argc, const char *const argv[])
{
    const programControl programCtrl(argc, argv);

    // Set cuda device
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipSetDevice(programCtrl.deviceList()[0]));
    checkCudaErrors(hipDeviceSynchronize());

    const host::latticeMesh mesh(programCtrl);

    VelocitySet::print();

    // Setup Streams
    const std::array<hipStream_t, 1> streamsLBM = host::createCudaStream();

    // Allocate the arrays on the host first
    const host::array<scalar_t, VelocitySet> h_rho("rho", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_u("u", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_v("v", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_w("w", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_m_xx("m_xx", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_m_xy("m_xy", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_m_xz("m_xz", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_m_yy("m_yy", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_m_yz("m_yz", mesh, programCtrl);
    const host::array<scalar_t, VelocitySet> h_m_zz("m_zz", mesh, programCtrl);

    device::array<scalar_t> rho(h_rho, mesh);
    device::array<scalar_t> u(h_u, mesh);
    device::array<scalar_t> v(h_v, mesh);
    device::array<scalar_t> w(h_w, mesh);
    device::array<scalar_t> mxx(h_m_xx, mesh);
    device::array<scalar_t> mxy(h_m_xy, mesh);
    device::array<scalar_t> mxz(h_m_xz, mesh);
    device::array<scalar_t> myy(h_m_yy, mesh);
    device::array<scalar_t> myz(h_m_yz, mesh);
    device::array<scalar_t> mzz(h_m_zz, mesh);

    const device::ptrCollection<10, scalar_t> devPtrs(
        rho.ptr(),
        u.ptr(),
        v.ptr(),
        w.ptr(),
        mxx.ptr(),
        mxy.ptr(),
        mxz.ptr(),
        myy.ptr(),
        myz.ptr(),
        mzz.ptr());

    device::halo<VelocitySet> blockHalo(
        {h_rho.arr(),
         h_u.arr(),
         h_v.arr(),
         h_w.arr(),
         h_m_xx.arr(),
         h_m_xy.arr(),
         h_m_xz.arr(),
         h_m_yy.arr(),
         h_m_yz.arr(),
         h_m_zz.arr()},
        mesh);

    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(momentBasedD3Q19), hipFuncCachePreferShared));

    const runTimeIO IO(mesh, programCtrl);

    for (label_t timeStep = programCtrl.latestTime(); timeStep < programCtrl.nt(); timeStep++)
    {
        // Do the run-time IO
        if (programCtrl.print(timeStep))
        {
            std::cout << "Time: " << timeStep << std::endl;
        }

        // Checkpoint
        // if (programCtrl.save(timeStep))
        // {
        //     fileIO::writeFile(
        //         programCtrl.caseName() + "_" + std::to_string(timeStep) + ".LBMBin",
        //         mesh,
        //         {"rho", "u", "v", "w", "m_xx", "m_xy", "m_xz", "m_yy", "m_yz", "m_zz"},
        //         host::toHost(devPtrs, mesh),
        //         timeStep);
        // }

        // Main kernel
        momentBasedD3Q19<<<mesh.gridBlock(), mesh.threadBlock(), 0, streamsLBM[0]>>>(
            devPtrs,
            blockHalo.fGhost(),
            blockHalo.gGhost());

        // Halo pointer swap
        blockHalo.swap();
    }

    return 0;
}