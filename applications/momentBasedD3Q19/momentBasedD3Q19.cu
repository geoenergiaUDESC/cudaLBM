#include "momentBasedD3Q19.cuh"

using namespace LBM;

int main(const int argc, const char *const argv[])
{
    const programControl programCtrl(argc, argv);

    // Set cuda device
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipSetDevice(programCtrl.deviceList()[0]));
    checkCudaErrors(hipDeviceSynchronize());

    const host::latticeMesh mesh(programCtrl);

    VSet::print();

    // Setup Streams
    const std::array<hipStream_t, 1> streamsLBM = host::createCudaStream();

    // Allocate the arrays on the host first
    const host::array<scalar_t, VSet> h_rho("rho", mesh, programCtrl);
    const host::array<scalar_t, VSet> h_u("u", mesh, programCtrl);
    const host::array<scalar_t, VSet> h_v("v", mesh, programCtrl);
    const host::array<scalar_t, VSet> h_w("w", mesh, programCtrl);
    const host::array<scalar_t, VSet> h_m_xx("m_xx", mesh, programCtrl);
    const host::array<scalar_t, VSet> h_m_xy("m_xy", mesh, programCtrl);
    const host::array<scalar_t, VSet> h_m_xz("m_xz", mesh, programCtrl);
    const host::array<scalar_t, VSet> h_m_yy("m_yy", mesh, programCtrl);
    const host::array<scalar_t, VSet> h_m_yz("m_yz", mesh, programCtrl);
    const host::array<scalar_t, VSet> h_m_zz("m_zz", mesh, programCtrl);

    device::array<scalar_t> rho(h_rho, mesh);
    device::array<scalar_t> u(h_u, mesh);
    device::array<scalar_t> v(h_v, mesh);
    device::array<scalar_t> w(h_w, mesh);
    device::array<scalar_t> mxx(h_m_xx, mesh);
    device::array<scalar_t> mxy(h_m_xy, mesh);
    device::array<scalar_t> mxz(h_m_xz, mesh);
    device::array<scalar_t> myy(h_m_yy, mesh);
    device::array<scalar_t> myz(h_m_yz, mesh);
    device::array<scalar_t> mzz(h_m_zz, mesh);

    const device::ptrCollection<10, scalar_t> devPtrs(
        rho.ptr(),
        u.ptr(),
        v.ptr(),
        w.ptr(),
        mxx.ptr(),
        mxy.ptr(),
        mxz.ptr(),
        myy.ptr(),
        myz.ptr(),
        mzz.ptr());

    device::halo<VSet> blockHalo(
        {h_rho.arr(),
         h_u.arr(),
         h_v.arr(),
         h_w.arr(),
         h_m_xx.arr(),
         h_m_xy.arr(),
         h_m_xz.arr(),
         h_m_yy.arr(),
         h_m_yz.arr(),
         h_m_zz.arr()},
        mesh);

    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(momentBasedD3Q19), hipFuncCachePreferShared));

    const runTimeIO IO(mesh, programCtrl);

    for (label_t timeStep = programCtrl.latestTime(); timeStep < programCtrl.nt(); timeStep++)
    {
        // Do the run-time IO
        if (programCtrl.print(timeStep))
        {
            std::cout << "Time: " << timeStep << std::endl;
        }

        // Checkpoint
        if (programCtrl.save(timeStep))
        {
            fileIO::writeFile(
                programCtrl.caseName() + "_" + std::to_string(timeStep) + ".LBMBin",
                mesh,
                {"rho", "u", "v", "w", "m_xx", "m_xy", "m_xz", "m_yy", "m_yz", "m_zz"},
                host::toHost(devPtrs, mesh),
                timeStep);
        }

        // Main kernel
        momentBasedD3Q19<<<mesh.gridBlock(), mesh.threadBlock(), 0, streamsLBM[0]>>>(
            devPtrs,
            blockHalo.fGhost(),
            blockHalo.gGhost());

        // Halo pointer swap
        blockHalo.swap();
    }

    return 0;
}